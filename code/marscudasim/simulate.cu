#include "hip/hip_runtime.h"
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>

#include "simulate.h"
#include "constants.h"



__device__
inline double get_B_phi(double R, double theta, double phidot)
{
    return R * R * sin(theta) * sin(theta) * phidot;
}

__device__
inline double get_B_theta(double R, double thetadot)
{
    return R * R * thetadot;
}

// from position and momentum vectors, returns generalized momentum, nondimensionalized
__device__
inline double get_B_R(double Rdot)
{
    return Rdot;
}

__device__
inline double get_Bdot_phi(double R, double theta, double phi, double* R_ks, double* theta_ks, double* phi_ks)
{
    return 0 //TODO: implement. it's a bit hairy.;
}

__device__
inline double get_Bdot_theta(double R, double theta, double phi, double B_phi, double* R_ks, double* theta_ks, double* phi_ks)
{
    return 0 //TODO: implement ;
}

__device__
inline double get_Bdot_R(double R, double theta, double phi, double B_theta, double B_phi, double* R_ks, double* theta_ks, double* phi_ks)
{
    return 0 //TODO: implement ;
}

__device__
inline double get_Rdot(double B_R):
{
    return B_R;
}

__device__
inline double get_thetadot(double R, double B_theta):
{
    return B_theta / (R * R);
}

__device__
inline double get_phidot(double R, double theta, double B_phi):
{
    return B_phi / (R * R * sin(theta) * sin(theta));
}

double * get_ephemerides_on_day(double * ephemerides, int ephsize, double day_idx){
    /*  input:
            per-day ephemerides for a given body, 
            the size of the coordinate set (column count, basically),
            and the non-integer day you want to interpolate on.
        output: 
            a list of interpolated coordinates for the given day
    */ 
    double day = day_idx + 1;

    int day_lb = int(floor(day));
    int day_ub = int(ceil(day));
    double day_diff = day % 1;

    double * lowerbound_eph = ephemerides[day_lb*ephsize]; // maybe needs sizeof(double) in the index??
    double * upperbound_eph = ephemerides[day_ub*ephsize]; // maybe needs sizeof(double) in the index??
    double result[ephsize] = 
    for(int i = 0; i < ephsize; i++){
        double lb_val = lowerbound_eph[i];
        double ub_val = upperbound_eph[i];

        double diff_val = ub_val - lb_val;
        result[i] = lb_val + diff_val * day_diff;
    }   
    return result;
}

// ----------------- MAIN ALGORITHM ----------------------------
// explicit euler algorithm for 4-body case
// All values are with nondimensionalized units
__device__
void simulate(psitype* psi,
                double h, // timestep
                double* score, 
                bool* success,
                int max_duration,
                int max_iter)
{
    // Unpack psi
    t = psi.t;
    Q = psi.Q;
    B = psi.B;
    burn = psi.burn;

    day = t* UNIT_TIME / (3600 * 24);

    ephemerides = get_ephemerides(max_year); //TODO: implement get_ephemerides for C

    R = Q[0];
    theta = Q[1];
    phi = Q[2];

    B_R = B[0];
    B_theta = B[1];
    B_phi = B[2];


    // ------ BEGIN SIMULATION ----
    i = 0;
    while(true)
    {
        i += 1;
        t += h;
        day = t * UNIT_TIME / (3600 * 24);

        eph_on_day = get_ephemerides_on_day(ephemerides, day);
        eph_coords = get_ephemerides_on_day_rad(ephemerides, day);

        euler_sym = euler_step_symplectic(h,Q,B,eph_coords);
        Q = euler_sym[0];
        B = euler_sym[1];

        if(t >= max_duration){
            print(".");
            break;
        }
        if( i >= max_iter){
            print('|');
            break;
        }
        if(false){
            // TODO: success handling
        }
    }
    score[0] = 100
    success[0] = false
}
