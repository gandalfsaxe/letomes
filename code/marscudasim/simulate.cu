#include "hip/hip_runtime.h"
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>

#include "simulate.h"
#include "constants.h"



__device__
inline double get_B_phi(double R, double theta, double phidot)
{
    return R * R * sin(theta) * sin(theta) * phidot;
}

__device__
inline double get_B_theta(double R, double thetadot)
{
    return R * R * thetadot;
}

// from position and momentum vectors, returns generalized momentum, nondimensionalized
__device__
inline double get_B_R(double Rdot)
{
    return Rdot;
}

__device__
inline double get_Bdot_phi(double R, double theta, double phi, double* R_ks, double* theta_ks, double* phi_ks)
{
    return 0 //TODO: implement. it's a bit hairy.;
}

__device__
inline double get_Bdot_theta(double R, double theta, double phi, double B_phi, double* R_ks, double* theta_ks, double* phi_ks)
{
    return 0 //TODO: implement ;
}

__device__
inline double get_Bdot_R(double R, double theta, double phi, double B_theta, double B_phi, double* R_ks, double* theta_ks, double* phi_ks)
{
    return 0 //TODO: implement ;
}

__device__
inline double get_Rdot(double B_R):
{
    return B_R;
}

__device__
inline double get_thetadot(double R, double B_theta):
{
    return B_theta / (R * R);
}

__device__
inline double get_phidot(double R, double theta, double B_phi):
{
    return B_phi / (R * R * sin(theta) * sin(theta));
}

// explicit euler algorithm for 4-body case
// All values are with nondimensionalized units
__device__
void simulate(psitype* psi,
                double h, // timestep
                double* score, 
                bool* success,
                int max_duration,
                int max_iter)
{
    // Unpack psi
    t = psi.t;
    Q = psi.Q;
    B = psi.B;
    burn = psi.burn;

    day = t* UNIT_TIME / (3600 * 24);

    ephemerides = get_ephemerides(max_year); //TODO: implement get_ephemerides for C

    R = Q[0];
    theta = Q[1];
    phi = Q[2];

    B_R = B[0];
    B_theta = B[1];
    B_phi = B[2];


    // ------ BEGIN SIMULATION ----
    i = 0;
    while(true)
    {
        i += 1;
        t += h;
        day = t * UNIT_TIME / (3600 * 24);

        eph_on_day = get_ephemerides_on_day(ephemerides, day);
        eph_coords = get_ephemerides_on_day_rad(ephemerides, day);

        euler_sym = euler_step_symplectic(h,Q,B,eph_coords);
        Q = euler_sym[0];
        B = euler_sym[1];

        if(t >= max_duration){
            print(".");
            break;
        }
        if( i >= max_iter){
            print('|');
            break;
        }
        if(false){
            // TODO: success handling
        }
    }
    score[0] = 100
    success[0] = false
}
