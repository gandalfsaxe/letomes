#include "hip/hip_runtime.h"
#include "constants.h"
#include "coordinate_system.h"
#include "equations_of_motion.h"

#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>

#define M_2PI (2.0 * M_PI)
#define COND false //i % 1 == 0

#include "euler_step.h"

extern "C" {

    __global__
    void simulate_kernel(int number_of_paths,
                         double h,
                         double max_duration,
                         int max_iter,
                         double* ts,
                         double* Rs, double* thetas, double* phis,
                         double* B_Rs, double* B_thetas, double* B_phis,
                         double* earth_R, double* earth_theta, double* earth_phi,
                         double* mars_R, double* mars_theta, double* mars_phi,
                         double* arive,
                         double* score)
    {
        const int pathNo = blockIdx.x * blockDim.x + threadIdx.x;
        if (pathNo < number_of_paths)
        {
            double t = ts[pathNo];
            double R = Rs[pathNo];
            double theta = thetas[pathNo];
            double phi = phis[pathNo];
            double B_R = B_Rs[pathNo];
            double B_theta = B_thetas[pathNo];
            double B_phi = B_phis[pathNo];
            double Rdot = get_Rdot(B_R);
            double thetadot = get_thetadot(R, B_theta);
            double phidot = get_phidot(R, theta, B_phi);
            //printf("pathNo=%i t=%.15e | R, theta, phi = [%.15e, %.15e, %.15e], B_R, B_theta, B_pi = [%.15e, %.15e, %.15e], Rdot, thetadot, pidot = [%.15e, %.15e, %.15e]\n", pathNo, t, R, theta, phi, B_R, B_theta, B_phi, Rdot, thetadot, phidot);
            // ------ BEGIN SIMULATION ----
            max_duration += t;
            int i = 0;
            double sun_distance, earth_distance, mars_distance, mars_orbit;
            double min_mars_distance = 1e200, min_day = 0;
            double day;
            while (true)
            {
                //if (COND) printf("i=%i | t=%.15lf | h=%.15lf | R=%.15lf, theta=%.15lf, phi=%.15lf | B_R=%.15lf, B_theta=%.15lf, B_phi=%.15lf\n", i, t, h, R, theta, phi, B_R, B_theta, B_phi);
                day = t * UNIT_TIME / (3600.0 * 24.0);
                int idx = day;
                double d = day - idx;
                idx++;
                double R_sun = SUN_R, theta_sun = SUN_THETA * M_PI / 180.0, phi_sun = SUN_PHI;
                double R_earth = lerp(earth_R[idx], earth_R[idx + 1], d);
                double theta_earth = lerp(earth_theta[idx], earth_theta[idx + 1], d);
                double phi_earth = lerp(earth_phi[idx], earth_phi[idx + 1], d);
                double R_mars = lerp(mars_R[idx], mars_R[idx + 1], d);
                double theta_mars = lerp(mars_theta[idx], mars_theta[idx + 1], d);
                double phi_mars = lerp(mars_phi[idx], mars_phi[idx + 1], d);
                sun_distance = UNIT_LENGTH *
                    distance(R, theta, phi, R_sun, theta_sun, phi_sun);
                earth_distance = UNIT_LENGTH *
                    distance(R, theta, phi, R_earth, theta_earth, phi_earth);
                mars_distance = UNIT_LENGTH *
                    distance(R, theta, phi, R_mars, theta_mars, phi_mars);
                mars_orbit = UNIT_LENGTH *
                    distance(R_sun, theta_sun, phi_sun, R_mars, theta_mars, phi_mars);
                min_day = min_mars_distance > mars_distance ?
                    day : min_day;
                min_mars_distance = min_mars_distance > mars_distance ?
                    mars_distance : min_mars_distance;
                if (false && i == 0)
                {
                    const char inside[] = "<=";
                    const char outside[] = "=>";
                    const char* away = sun_distance < mars_orbit ? inside : outside;
                    printf("duration=%f i=%i pathNo=%i | R=%.15f theta=%.15f phi=%.15f | sun_dist=%i km earth_dist=%i km mars_dist=%i km | mars_orbit=%i km %s\n",
                           (t - ts[pathNo]) * UNIT_TIME / (3600.0 * 24.0),
                           i, pathNo, R, theta, phi,
                           (int)sun_distance,
                           (int)earth_distance,
                           (int)mars_distance,
                           (int)mars_orbit,
                           away);
                    //printf("day=%.15e idx=%i d=%.15e\n", day, idx, d);
                    //printf("R_sun=%.15lf, theta_sun=%.15lf, phi_sun=%.15lf\n", R_sun, theta_sun, phi_sun);
                    //printf("R_earth=%.15lf, theta_earth=%.15lf, phi_earth=%.15lf\n", R_earth, theta_earth, phi_earth);
                    //printf("R_mars=%.15lf, theta_mars=%.15lf, phi_mars=%.15lf\n", R_mars, theta_mars, phi_mars);
                }
                if (t >= max_duration) {
                    printf(".");
                    arive[pathNo] = min_day;
                    score[pathNo] = min_mars_distance;
                    break;
                }
                if (i >= max_iter) {
                    printf("|");
                    arive[pathNo] = min_day;
                    score[pathNo] = min_mars_distance;
                    break;
                }
                if (sun_distance <= SUN_RADIUS)
                {
                    printf("*");
                    arive[pathNo] = min_day;
                    score[pathNo] = min_mars_distance;
                    break;
                }
                if (earth_distance <= EARTH_RADIUS)
                {
                    printf("o");
                    arive[pathNo] = min_day;
                    score[pathNo] = min_mars_distance;
                    break;
                }
                if (mars_distance <= MARS_RADIUS)
                {
                    printf("X");
                    arive[pathNo] = min_day;
                    score[pathNo] = 0;
                    break;
                }
                /*
                if (mars_distance > min_mars_distance)
                {
                    printf("-");
                    arive[pathNo] = day;
                    score[pathNo] = min_mars_distance;
                    break;
                }
                */
                euler_step(h, R, theta, phi, B_R, B_theta, B_phi,
                           R_sun, theta_sun, phi_sun,
                           R_earth, theta_earth, phi_earth,
                           R_mars, theta_mars, phi_mars,
                           &R, &theta, &phi, &B_R, &B_theta, &B_phi);
                t += h;
                i += 1;
            }
            /*
            const char inside[] = "<=";
            const char outside[] = "=>";
            const char* away = sun_distance < mars_orbit ? inside : outside;
            printf("END: duration=%f i=%i pathNo=%i | R=%.15f theta=%.15f phi=%.15f | sun_dist=%i km earth_dist=%i km mars_dist=%i km | mars_orbit=%i km min_mars_dist=%i km %s\n",
                   (t - ts[pathNo]) * UNIT_TIME / (3600.0 * 24.0),
                   i, pathNo, R, theta, phi,
                   (int)sun_distance,
                   (int)earth_distance,
                   (int)mars_distance,
                   (int)mars_orbit,
                   (int)min_mars_distance,
                   away);
            */
        }
    }

    void deviceQuery(int devNo)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, devNo);
        printf("Device %i: \"%s\".\n", devNo, prop.name);
        int nProcessors = prop.multiProcessorCount;
        int nCores = _ConvertSMVer2Cores(prop.major, prop.minor) * nProcessors;
        int clockFreq = prop.clockRate / 1000;
        int peakPerformanceSP = round(2.0e-3 * clockFreq * nCores);
        int singleToDoubleRatio = prop.singleToDoublePrecisionPerfRatio;
        int peakPerformanceDP = peakPerformanceSP / singleToDoubleRatio;
        int peakBandwidth = round(2.0 * prop.memoryClockRate * 1e-6 *
                                  (prop.memoryBusWidth / 8));
        printf("  Multiprocessors:  %6i\n", nProcessors);
        printf("  Cores:            %6i\n", nCores);
        printf("  Peak performance: %6i GFlops\n", peakPerformanceDP);
        printf("  Peak bandwidth:   %6i GB/s\n", peakBandwidth);
    }

    void deviceWarmup(int devNo)
    {
        printf("Warming up device: "); fflush(stdout);
        double time = omp_get_wtime();
        hipSetDevice(devNo);
        double *dummy_d;
        hipMalloc((void**)&dummy_d, 0); // We force the creation of context on
                                         // the device by allocating a dummy.
        printf("%6.4f seconds\n", omp_get_wtime() - time);
    }

    void simulate(int number_of_paths,
                  double h,
                  double max_duration,
                  int max_iter,
                  double* t,
                  double* R, double* theta, double* phi,
                  double* B_R, double* B_theta, double* B_phi,
                  int size_ephemerides,
                  double* earth_R, double* earth_theta, double* earth_phi,
                  double* mars_R, double* mars_theta, double* mars_phi,
                  double* arive,
                  double* score)
    {
        printf("cudasim::simulate(number_of_paths=%i, h=%.15le, max_duration=%.15le, max_iter=%i\n", number_of_paths, h, max_duration, max_iter);

        // Warm up..
        const int devNo = 0;
        deviceQuery(devNo);
        deviceWarmup(devNo);

        // Allocate memory.
        double time = omp_get_wtime();
        printf("Allocate memory + transfer ephemerides: ");
        const int nBytes = number_of_paths * sizeof(double);
        const int nBytesArive = number_of_paths * sizeof(double);
        const int nBytesEphemerides = size_ephemerides * sizeof(double);
        double* t_d;
        double* R_d;
        double* theta_d;
        double* phi_d;
        double* B_R_d;
        double* B_theta_d;
        double* B_phi_d;
        double* arive_d;
        double* score_d;
        double* earth_R_d;
        double* earth_theta_d;
        double* earth_phi_d;
        double* mars_R_d;
        double* mars_theta_d;
        double* mars_phi_d;
        checkCudaErrors(hipMalloc((void**)&t_d, nBytes));
        checkCudaErrors(hipMalloc((void**)&R_d, nBytes));
        checkCudaErrors(hipMalloc((void**)&theta_d, nBytes));
        checkCudaErrors(hipMalloc((void**)&phi_d, nBytes));
        checkCudaErrors(hipMalloc((void**)&B_R_d, nBytes));
        checkCudaErrors(hipMalloc((void**)&B_theta_d, nBytes));
        checkCudaErrors(hipMalloc((void**)&B_phi_d, nBytes));
        checkCudaErrors(hipMalloc((void**)&earth_R_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&earth_theta_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&earth_phi_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&mars_R_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&mars_theta_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&mars_phi_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&arive_d, nBytesArive));
        checkCudaErrors(hipMalloc((void**)&score_d, nBytes));
        checkCudaErrors(hipMemcpy(t_d,
                                   t,
                                   nBytes,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(R_d,
                                   R,
                                   nBytes,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(theta_d,
                                   theta,
                                   nBytes,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(phi_d,
                                   phi,
                                   nBytes,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(B_R_d,
                                   B_R,
                                   nBytes,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(B_theta_d,
                                   B_theta,
                                   nBytes,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(B_phi_d,
                                   B_phi,
                                   nBytes,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(earth_R_d,
                                   earth_R,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(earth_theta_d,
                                   earth_theta,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(earth_phi_d,
                                   earth_phi,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(mars_R_d,
                                   mars_R,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(mars_theta_d,
                                   mars_theta,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(mars_phi_d,
                                   mars_phi,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        printf("%6.4f seconds\n", omp_get_wtime() - time);

        // Launch kernel.
        time = omp_get_wtime();
        printf("Running kernel: \n");
        dim3 block(160);
        dim3 grid((number_of_paths - 1) / block.x + 1);
        simulate_kernel<<<grid, block>>>(number_of_paths,
                                         h,
                                         max_duration,
                                         max_iter,
                                         t_d,
                                         R_d, theta_d, phi_d,
                                         B_R_d, B_theta_d, B_phi_d,
                                         earth_R_d, earth_theta_d, earth_phi_d,
                                         mars_R_d, mars_theta_d, mars_phi_d,
                                         arive_d,
                                         score_d);
        checkCudaErrors(hipDeviceSynchronize());
        printf("%6.4f seconds\n", omp_get_wtime() - time);

        // Copy results.
        time = omp_get_wtime();
        printf("Transfering result (%i KiB): ", (nBytesArive + nBytes) / 1024);
        checkCudaErrors(hipMemcpy(arive,
                                   arive_d,
                                   nBytesArive,
                                   hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(score,
                                   score_d,
                                   nBytes,
                                   hipMemcpyDeviceToHost));
        printf("%6.4f seconds\n", omp_get_wtime() - time);

        // Clean up.
        checkCudaErrors(hipFree(arive_d));
        checkCudaErrors(hipFree(score_d));
    }

} // extern "C"
