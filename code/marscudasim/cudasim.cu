#include "hip/hip_runtime.h"
#include "constants.h"
//#include "simulate.h"

#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>

//#define M_PI 3.14159265358979
#define M_2PI (2.0 * M_PI)

__device__
inline double get_Rdot(double B_R)
{
    return B_R;
}

__device__
inline double get_thetadot(double R, double B_theta)
{
    return B_theta / (R * R);
}

__device__
inline double get_phidot(double R, double theta, double B_phi)
{
    return B_phi / (R * R * sin(theta) * sin(theta));
}

__device__
inline double get_B_R(double Rdot)
{
    return Rdot;
}

__device__
inline double get_B_theta(double R, double thetadot)
{
    return R * R * thetadot;
}

__device__
inline double get_B_phi(double R, double theta, double phidot)
{
    return R * R * sin(theta) * sin(theta) * phidot;
}

__device__
inline double get_Bdot_R(double R, double theta, double phi, 
                         double B_theta, double B_phi,
                         double R_sun, double theta_sun, double phi_sun,
                         double R_earth, double theta_earth, double phi_earth,
                         double R_mars, double theta_mars, double phi_mars)
{
    double numerator_sun = SUN_ETA *
        (-R + R_sun * (cos(theta) * cos(theta_sun) + sin(theta) * sin(theta_sun) * cos(phi - phi_sun)));
    double denominator_sun = R * R + R_sun * R_sun -
        2.0 * R * R_sun * (cos(theta) * cos(theta_sun) +
                           sin(theta) * sin(theta_sun) * cos(phi - phi_sun));
    denominator_sun = denominator_sun * sqrt(denominator_sun);

    double numerator_earth = EARTH_ETA *
        (-R + R_earth * (cos(theta) * cos(theta_earth) + sin(theta) * sin(theta_earth) * cos(phi - phi_earth)));
    double denominator_earth = R * R + R_earth * R_earth -
        2.0 * R * R_earth * (cos(theta) * cos(theta_earth) +
                             sin(theta) * sin(theta_earth) * cos(phi - phi_earth));
    denominator_earth = denominator_earth * sqrt(denominator_earth);

    double numerator_mars = MARS_ETA *
        (-R + R_mars * (cos(theta) * cos(theta_mars) + sin(theta) * sin(theta_mars) * cos(phi - phi_mars)));
    double denominator_mars = R * R + R_mars * R_mars -
        2.0 * R * R_mars * (cos(theta) * cos(theta_mars) +
                            sin(theta) * sin(theta_mars) * cos(phi - phi_mars));
    denominator_mars = denominator_mars * sqrt(denominator_mars);
    double Bdot_R1 = (B_theta * B_theta) / (R * R * R);
    double Bdot_R2 = (B_phi * B_phi) /(R * R * R * sin(theta) * sin(theta));
    double Bdot_R3 =
        (numerator_sun / denominator_sun) +
        (numerator_earth / denominator_earth) +
        (numerator_mars / denominator_mars);
    /*
    printf("====== numerator_sun=%.15le numerator_earth=%.15le numerator_mars=%.15le\n", numerator_sun, numerator_earth, numerator_mars);
    printf("====== denominator_sun=%.15le denominator_earth=%.15le denominator_mars=%.15le\n", denominator_sun, denominator_earth, denominator_mars);
    printf("====== Bdot_R1=%.15lf Bdot_R2=%.15lf Bdot_R3=%.15lf\n", Bdot_R1, Bdot_R2, Bdot_R3);
    */
    return Bdot_R1 + Bdot_R2 + Bdot_R3;
}

__device__
inline double get_Bdot_theta(double R, double theta, double phi, 
                             double B_phi,
                             double R_sun, double theta_sun, double phi_sun,
                             double R_earth, double theta_earth, double phi_earth,
                             double R_mars, double theta_mars, double phi_mars)
{
    double numerator_sun = SUN_ETA * 
        (R * R_sun * (-sin(theta) * cos(theta_sun) + cos(theta) * sin(theta_sun) * cos(phi - phi_sun)));
    double denominator_sun = R * R + R_sun * R_sun -
        2.0 * R * R_sun * (cos(theta) * cos(theta_sun) +
                           sin(theta) * sin(theta_sun) * cos(phi - phi_sun));
    denominator_sun = denominator_sun * sqrt(denominator_sun);
    double numerator_earth = EARTH_ETA * 
        (R * R_earth * (-sin(theta) * cos(theta_earth) + cos(theta) * sin(theta_earth) * cos(phi - phi_earth)));
    double denominator_earth = R * R + R_earth * R_earth -
        2.0 * R * R_earth * (cos(theta) * cos(theta_earth) +
                           sin(theta) * sin(theta_earth) * cos(phi - phi_earth));
    denominator_earth = denominator_earth * sqrt(denominator_earth);
    double numerator_mars = MARS_ETA * 
        (R * R_mars * (-sin(theta) * cos(theta_mars) + cos(theta) * sin(theta_mars) * cos(phi - phi_mars)));
    double denominator_mars = R * R + R_mars * R_mars -
        2.0 * R * R_mars * (cos(theta) * cos(theta_mars) +
                           sin(theta) * sin(theta_mars) * cos(phi - phi_mars));
    denominator_mars = denominator_mars * sqrt(denominator_mars);

    double Bdot_theta1 = (B_phi * B_phi) /
        (R * R * sin(theta) * sin(theta) * tan(theta));
    double Bdot_theta2 =
        numerator_sun / denominator_sun + 
        numerator_earth / denominator_earth + 
        numerator_mars / denominator_mars;
    return Bdot_theta1 + Bdot_theta2;
}

__device__
inline double get_Bdot_phi(double R, double theta, double phi, 
                           double R_sun, double theta_sun, double phi_sun,
                           double R_earth, double theta_earth, double phi_earth,
                           double R_mars, double theta_mars, double phi_mars)
{
    double numerator_sun = SUN_ETA * 
        (-R * R_sun * sin(theta) * sin(theta_sun) * sin(phi - phi_sun));
    double denominator_sun = R * R + R_sun * R_sun -
        2.0 * R * R_sun * (cos(theta) * cos(theta_sun) +
                           sin(theta) * sin(theta_sun) * cos(phi - phi_sun));
    denominator_sun = denominator_sun * sqrt(denominator_sun);
    double numerator_earth = EARTH_ETA * 
        (-R * R_earth * sin(theta) * sin(theta_earth) * sin(phi - phi_earth));
    double denominator_earth = R * R + R_earth * R_earth -
        2.0 * R * R_earth * (cos(theta) * cos(theta_earth) +
                           sin(theta) * sin(theta_earth) * cos(phi - phi_earth));
    denominator_earth = denominator_earth * sqrt(denominator_earth);
    double numerator_mars = MARS_ETA * 
        (-R * R_mars * sin(theta) * sin(theta_mars) * sin(phi - phi_mars));
    double denominator_mars = R * R + R_mars * R_mars -
        2.0 * R * R_mars * (cos(theta) * cos(theta_mars) +
                           sin(theta) * sin(theta_mars) * cos(phi - phi_mars));
    denominator_mars = denominator_mars * sqrt(denominator_mars);
    return
        numerator_sun / denominator_sun + 
        numerator_earth / denominator_earth + 
        numerator_mars / denominator_mars;
}

__device__
inline double keep_theta_in_interval_zero_to_pi(double theta)
{
    theta = fmod(theta,  M_2PI);
    return theta > M_PI ? M_2PI - theta : theta;
}

__device__
inline double keep_phi_in_interval_npi_to_pi(double phi)
{
    phi = fmod(phi, M_2PI);
    return phi > M_PI ? phi - M_2PI : phi;
}

__device__
inline double lerp(double v0, double v1, double t)
{
    return (1 - t) * v0 + t * v1;
    //return v0 + t * (v1 - v0);
}

extern "C" {


    // ----------------- MAIN ALGORITHM ----------------------------
    // explicit euler algorithm for 4-body case
    // All values are with nondimensionalized units
    __global__
    void simulate_kernel(int number_of_paths,
                         double fan_delta,
                         int coordinate_no,
                         double h,
                         double max_duration,
                         int max_iter,
                         double t,
                         double R, double theta, double phi,
                         double B_R, double B_theta, double B_phi,
                         double* earth_R, double* earth_theta, double* earth_phi,
                         double* mars_R, double* mars_theta, double* mars_phi,
                         bool* success,
                         double* score)
    {
        const int pathNo = blockIdx.x * blockDim.x + threadIdx.x;
        if (pathNo < number_of_paths)
        {
            // Implement simple fan search.
            double dB = (fan_delta / number_of_paths) * pathNo -0.5 * fan_delta;
            B_R = coordinate_no == 0 ? B_R + dB : B_R;
            B_theta = coordinate_no == 1 ? B_theta + dB : B_theta;
            B_phi = coordinate_no == 2 ? B_phi + dB : B_phi;

            // ------ BEGIN SIMULATION ----
            int i = 0;
            while (true)
            {
                //if (i % 1000 == 0) printf("i=%i | t=%.15lf | h=%.15lf | R=%.15lf, theta=%.15lf, phi=%.15lf | B_R=%.15lf, B_theta=%.15lf, B_phi=%.15lf\n", i, t, h, R, theta, phi, B_R, B_theta, B_phi);
                i += 1;
                t += h;

                double day = t * UNIT_TIME / (3600.0 * 24.0);
                int idx = day;
                double d = day - idx;
                idx++;
                double R_sun = SUN_R, theta_sun = SUN_THETA * M_PI / 180.0, phi_sun = SUN_PHI;
                double R_earth = lerp(earth_R[idx], earth_R[idx + 1], d);
                double theta_earth = lerp(earth_theta[idx], earth_theta[idx + 1], d);
                double phi_earth = lerp(earth_phi[idx], earth_phi[idx + 1], d);
                double R_mars = lerp(mars_R[idx], mars_R[idx + 1], d);
                double theta_mars = lerp(mars_theta[idx], mars_theta[idx + 1], d);
                double phi_mars = lerp(mars_phi[idx], mars_phi[idx + 1], d);
                //printf("R_sun=%.15lf, theta_sun=%.15lf, phi_sun=%.15lf\n", R_sun, theta_sun, phi_sun);
                //printf("R_earth=%.15lf, theta_earth=%.15lf, phi_earth=%.15lf\n", R_earth, theta_earth, phi_earth);
                //printf("R_mars=%.15lf, theta_mars=%.15lf, phi_mars=%.15lf\n", R_mars, theta_mars, phi_mars);

                // Update q
                R = R + h * get_Rdot(B_R);
                theta = theta + h * get_thetadot(R, B_theta);
                phi = phi + h * get_phidot(R, theta, B_phi);
                theta = keep_theta_in_interval_zero_to_pi(theta);
                phi = keep_phi_in_interval_npi_to_pi(phi);

                // Update B_R
                B_R += h * get_Bdot_R(R, theta, phi,
                                      B_theta, B_phi,
                                      R_sun, theta_sun, phi_sun,
                                      R_earth, theta_earth, phi_earth,
                                      R_mars, theta_mars, phi_mars);
                // Update B_theta
                B_theta += h * get_Bdot_theta(R, theta, phi,
                                              B_phi,
                                              R_sun, theta_sun, phi_sun,
                                              R_earth, theta_earth, phi_earth,
                                              R_mars, theta_mars, phi_mars);
                // Update B_phi
                B_phi += h * get_Bdot_phi(R, theta, phi,
                                          R_sun, theta_sun, phi_sun,
                                          R_earth, theta_earth, phi_earth,
                                          R_mars, theta_mars, phi_mars);

                if (t >= max_duration) {
                    printf(".");
                    success[pathNo] = false;
                    return;
                }
                if (i >= max_iter) {
                    printf("|");
                    success[pathNo] = false;
                    return;
                }
            }
            printf("X");
            score[pathNo] = 100;
            success[pathNo] = true;
        }
    }

    void deviceQuery(int devNo)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, devNo);
        printf("Device %i: \"%s\".\n", devNo, prop.name);
        int nProcessors = prop.multiProcessorCount;
        int nCores = _ConvertSMVer2Cores(prop.major, prop.minor) * nProcessors;
        int clockFreq = prop.clockRate / 1000;
        int peakPerformanceSP = round(2.0e-3 * clockFreq * nCores);
        int singleToDoubleRatio = prop.singleToDoublePrecisionPerfRatio;
        int peakPerformanceDP = peakPerformanceSP / singleToDoubleRatio;
        int peakBandwidth = round(2.0 * prop.memoryClockRate * 1e-6 *
                                  (prop.memoryBusWidth / 8));
        printf("  Multiprocessors:  %6i\n", nProcessors);
        printf("  Cores:            %6i\n", nCores);
        printf("  Peak performance: %6i GFlops\n", peakPerformanceDP);
        printf("  Peak bandwidth:   %6i GB/s\n", peakBandwidth);
    }

    void deviceWarmup(int devNo)
    {
        printf("Warming up device: "); fflush(stdout);
        double time = omp_get_wtime();
        hipSetDevice(devNo);
        double *dummy_d;
        hipMalloc((void**)&dummy_d, 0); // We force the creation of context on
                                         // the device by allocating a dummy.
        printf("%6.4f seconds\n", omp_get_wtime() - time);
    }

    void simulate(int number_of_paths,
                  double fan_delta,
                  int coordinate_no,
                  double h,
                  double max_duration,
                  int max_iter,
                  double t,
                  double* Q,
                  double* B,
                  int size_ephemerides,
                  double* earth_R, double* earth_theta, double* earth_phi,
                  double* mars_R, double* mars_theta, double* mars_phi,
                  bool* success,
                  double* score)
    {
        printf("cudasim::simulate(number_of_paths=%i, fan_delta=%.15le, coordinate_no=%i, h=%.15le, max_duration=%.15le, max_iter=%i, t=%.15le, Q=[%f,%f,%f], B=[%f,%f,%f]\n", number_of_paths, fan_delta, coordinate_no, h, max_duration, max_iter, t, Q[0], Q[1], Q[2], B[0], B[1], B[2]);

        // Warm up..
        const int devNo = 1;
        deviceQuery(devNo);
        deviceWarmup(devNo);

        // Allocate memory.
        double time = omp_get_wtime();
        printf("Allocate memory + transfer ephemerides: ");
        const int nBytesScore = number_of_paths * sizeof(double);
        const int nBytesSuccess = number_of_paths * sizeof(bool);
        const int nBytesEphemerides = size_ephemerides * sizeof(double);
        bool* success_d;
        double* score_d;
        double* earth_R_d;
        double* earth_theta_d;
        double* earth_phi_d;
        double* mars_R_d;
        double* mars_theta_d;
        double* mars_phi_d;
        checkCudaErrors(hipMalloc((void**)&earth_R_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&earth_theta_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&earth_phi_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&mars_R_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&mars_theta_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&mars_phi_d, nBytesEphemerides));
        checkCudaErrors(hipMalloc((void**)&success_d, nBytesSuccess));
        checkCudaErrors(hipMalloc((void**)&score_d, nBytesScore));
        checkCudaErrors(hipMemcpy(earth_R_d,
                                   earth_R,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(earth_theta_d,
                                   earth_theta,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(earth_phi_d,
                                   earth_phi,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(mars_R_d,
                                   mars_R,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(mars_theta_d,
                                   mars_theta,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        checkCudaErrors(hipMemcpy(mars_phi_d,
                                   mars_phi,
                                   nBytesEphemerides,
                                   hipMemcpyHostToDevice));
        printf("%6.4f seconds\n", omp_get_wtime() - time);

        // Launch kernel.
        time = omp_get_wtime();
        printf("Running kernel: \n");
        dim3 block(256);
        dim3 grid((number_of_paths - 1) / block.x + 1);
        simulate_kernel<<<grid, block>>>(number_of_paths,
                                         fan_delta,
                                         coordinate_no,
                                         h,
                                         max_duration,
                                         max_iter,
                                         t,
                                         Q[0], Q[1], Q[2],
                                         B[0], B[1], B[2],
                                         earth_R_d, earth_theta_d, earth_phi_d,
                                         mars_R_d, mars_theta_d, mars_phi_d,
                                         success_d,
                                         score_d);
        checkCudaErrors(hipDeviceSynchronize());
        printf("%6.4f seconds\n", omp_get_wtime() - time);

        // Copy results.
        time = omp_get_wtime();
        printf("Transfering result (%i KiB): ", (nBytesSuccess + nBytesScore) / 1024);
        checkCudaErrors(hipMemcpy(success,
                                   success_d,
                                   nBytesSuccess,
                                   hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(score,
                                   score_d,
                                   nBytesScore,
                                   hipMemcpyDeviceToHost));
        printf("%6.4f seconds\n", omp_get_wtime() - time);

        // Clean up.
        checkCudaErrors(hipFree(success_d));
        checkCudaErrors(hipFree(score_d));
    }

} // extern "C"
