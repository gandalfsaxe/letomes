#include "hip/hip_runtime.h"
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>

#include "constants.h"
#include "simulate.h"

extern "C" {

    __global__
    void kernel(int nTrajectories,
                psitype* psis,
                bool* successes,
                double* scores)
    {
        const int trajIdx = blockIdx.x * blockDim.x + threadIdx.x;
        
        if trajIdx < nTrajectories //for each path
        {
            psitype* thisPsi = psis[sizeof(psitype) * trajIdx];
            simulate(thisPsi, &successes[trajIdx], &scores[trajIdx]);
        }
    }

    void deviceQuery(int devNo)
    {
        hipDeviceProp_t prop;
        hipGetDeviceProperties(&prop, devNo);
        printf("Device %i: \"%s\".\n", devNo, prop.name);
        int nProcessors = prop.multiProcessorCount;
        int nCores = _ConvertSMVer2Cores(prop.major, prop.minor) * nProcessors;
        int clockFreq = prop.clockRate / 1000;
        int peakPerformanceSP = round(2.0e-3 * clockFreq * nCores);
        int singleToDoubleRatio = prop.singleToDoublePrecisionPerfRatio;
        int peakPerformanceDP = peakPerformanceSP / singleToDoubleRatio;
        int peakBandwidth = round(2.0 * prop.memoryClockRate * 1e-6 *
                                  (prop.memoryBusWidth / 8));
        printf("  Multiprocessors:  %6i\n", nProcessors);
        printf("  Cores:            %6i\n", nCores);
        printf("  Peak performance: %6i GFlops\n", peakPerformanceDP);
        printf("  Peak bandwidth:   %6i GB/s\n", peakBandwidth);
    }

    void deviceWarmup(int devNo)
    {
        printf("Warming up device... "); fflush(stdout);
        double time = omp_get_wtime();
        hipSetDevice(devNo);
        double *dummy_d;
        hipMalloc((void**)&dummy_d, 0); // We force the creation of context on
                                         // the device by allocating a dummy.
        printf("%6.4f seconds\n", omp_get_wtime() - time);
    }

    // launch (not really a launch since we start from LEO) a 
    // single rocket with a given set of hyperparameters, return the path
    __global__
    void oldkernel(int nPaths,
                double maxDuration,
                int maxSteps,
                double* points,
                bool* success,
                double* score)
    {
        const int pathNo = blockIdx.x * blockDim.x + threadIdx.x;

        if (pathNo < nPaths)
        {
            double posAng = points[3 * pathNo + 0];
            double burnAng = points[3 * pathNo + 1];
            double burnDv = points[3 * pathNo + 2];

            maxDuration /= UNIT_TIME;

            // position (where on earth do we start our burn)
            double x0 = cos(posAng) * LEO_RADIUS_NONDIM;
            double y0 = sin(posAng) * LEO_RADIUS_NONDIM;
            x0 += EARTH_POSITION_X;

            // how fast are we going when we start?
            double vhat_x = -sin(posAng);
            double vhat_y = cos(posAng);
            double v_x = LEO_VELOCITY_NONDIM * vhat_x;
            double v_y = LEO_VELOCITY_NONDIM * vhat_y;

            // burn vector: At what angle do we launch outward, and how hard do we push?
            double burnDv_x = cos(burnAng) * vhat_x - sin(burnAng) * vhat_y;
            double burnDv_y = sin(burnAng) * vhat_x + cos(burnAng) * vhat_y;

            // resultant momentum vector
            double p0_x = v_x + burnDv * burnDv_x / UNIT_VELOCITY - y0;
            double p0_y = v_y + burnDv * burnDv_y / UNIT_VELOCITY + x0;

            // SIMULATE
            //printf("pathNo = %i | posAng = %.15lf, burnAng = %.15lf, burnDv = %.15lf: running symplectic with [x0, y0, p0_x, p0_y] = {%.15lf, %.15lf, %.15lf, %.15lf}\n", pathNo, posAng, burnAng, burnDv, x0, y0, p0_x, p0_y);
            symplectic(x0,
                       y0,
                       p0_x,
                       p0_y,
                       maxDuration,
                       maxSteps,
                       &score[pathNo],
                       &success[pathNo]);
        }
    }

    void integrate(int nIndividuals,
                   int nJitter,
                   double maxDuration,
                   int maxSteps,
                   double* points,
                   bool* success,
                   double* score
                   )
    {
        const int devNo = 1;

        deviceQuery(devNo);
        deviceWarmup(devNo);

        // Allocate array on device and transfer input.
        bool* success_d; double* points_d, * score_d;
        const int nPaths = nIndividuals * nJitter;
        const int nBytesPath = nPaths * 3 * sizeof(double);
        const int nBytesSuccess = nPaths * sizeof(bool);
        const int nBytesScore = nPaths * sizeof(double);
        const int nBytes = nBytesPath + nBytesSuccess + nBytesScore;
        double time = omp_get_wtime();
        printf("Transfering input (%i KiB)... ", nBytes / 1024);
        checkCudaErrors(hipMalloc((void**)&points_d, nBytesPath));
        checkCudaErrors(hipMalloc((void**)&success_d, nBytesSuccess));
        checkCudaErrors(hipMalloc((void**)&score_d, nBytesScore));
        checkCudaErrors(hipMemcpy(points_d,
                                   points,
                                   nBytesPath,
                                   hipMemcpyHostToDevice));
        printf("%6.4f seconds\n", omp_get_wtime() - time);

        // Launch kernel.
        time = omp_get_wtime();
        printf("Running kernel... ");
        dim3 block(256);
        dim3 grid((nPaths - 1) / block.x + 1);
        kernel<<<grid, block>>>(nPaths, maxDuration, maxSteps, points_d, success_d, score_d);
        checkCudaErrors(hipDeviceSynchronize());
        printf("%6.4f seconds\n", omp_get_wtime() - time);

        time = omp_get_wtime();
        printf("Transfering result (%i KiB)... ", (nBytesSuccess + nBytesScore) / 1024);
        checkCudaErrors(hipMemcpy(success,
                                   success_d,
                                   nBytesSuccess,
                                   hipMemcpyDeviceToHost));
        checkCudaErrors(hipMemcpy(score,
                                   score_d,
                                   nBytesScore,
                                   hipMemcpyDeviceToHost));
        printf("%6.4f seconds\n", omp_get_wtime() - time);

        // Clean up.
        checkCudaErrors(hipFree(points_d));
        checkCudaErrors(hipFree(success_d));
        checkCudaErrors(hipFree(score_d));
    }
}
