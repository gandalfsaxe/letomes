#include "constants.h"
#include "coordinate_system.h"
#include "euler_step.h"

#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>

#define M_2PI (2.0 * M_PI)
#define COND false //i % 1 == 0

extern "C" {

    void simulate_cpu(double h,
                      double max_duration,
                      int max_iter,
                      double* ts,
                      double* Rs, double* thetas, double* phis,
                      double* B_Rs, double* B_thetas, double* B_phis,
                      int size_ephemerides,
                      double* earth_R, double* earth_theta, double* earth_phi,
                      double* mars_R, double* mars_theta, double* mars_phi,
                      double* ts_out,
                      double* Qs_out,
                      int* i_final)
    {
        printf("cudasim::simulate_single(h=%.15le, max_duration=%.15le, max_iter=%i\n", h, max_duration, max_iter);
        double time = omp_get_wtime();
        printf("Running kernel: \n");

        const int pathNo = 0;
        double t = ts[pathNo];
        double R = Rs[pathNo];
        double theta = thetas[pathNo];
        double phi = phis[pathNo];
        double B_R = B_Rs[pathNo];
        double B_theta = B_thetas[pathNo];
        double B_phi = B_phis[pathNo];
        double Rdot = get_Rdot(B_R);
        double thetadot = get_thetadot(R, B_theta);
        double phidot = get_phidot(R, theta, B_phi);
        // ------ BEGIN SIMULATION ----
        max_duration += t;
        int i = 0;
        double sun_distance, earth_distance, mars_distance, mars_orbit;
        double min_mars_distance = 1e200, min_day = 0;
        double day;
        while (true)
        {
            ts_out[i] = t;
            Qs_out[i * 3 + 0] = R;
            Qs_out[i * 3 + 1] = theta;
            Qs_out[i * 3 + 2] = phi;

            //if (i % 1 == 0) printf("i=%i | t=%.15lf | h=%.15lf | R=%.15lf, theta=%.15lf, phi=%.15lf | B_R=%.15lf, B_theta=%.15lf, B_phi=%.15lf\n", i, t, h, R, theta, phi, B_R, B_theta, B_phi);
            day = t * UNIT_TIME / (3600.0 * 24.0);
            int idx = day;
            double d = day - idx;
            idx++;
            double R_sun = SUN_R;
            double theta_sun = SUN_THETA * M_PI / 180.0;
            double phi_sun = SUN_PHI;
            double R_earth = lerp(earth_R[idx], earth_R[idx + 1], d);
            double theta_earth = lerp(earth_theta[idx], earth_theta[idx + 1], d);
            double phi_earth = lerp(earth_phi[idx], earth_phi[idx + 1], d);
            double R_mars = lerp(mars_R[idx], mars_R[idx + 1], d);
            double theta_mars = lerp(mars_theta[idx], mars_theta[idx + 1], d);
            double phi_mars = lerp(mars_phi[idx], mars_phi[idx + 1], d);

            sun_distance = UNIT_LENGTH *
                distance(R, theta, phi, R_sun, theta_sun, phi_sun);
            earth_distance = UNIT_LENGTH *
                distance(R, theta, phi, R_earth, theta_earth, phi_earth);
            mars_distance = UNIT_LENGTH *
                distance(R, theta, phi, R_mars, theta_mars, phi_mars);
            mars_orbit = UNIT_LENGTH *
                distance(R_sun, theta_sun, phi_sun, R_mars, theta_mars, phi_mars);
            min_day = min_mars_distance > mars_distance ?
                day : min_day;
            min_mars_distance = min_mars_distance > mars_distance ?
                mars_distance : min_mars_distance;

            if (i == 0 || i % 100000 == 0)
            {
                printf("duration=%f i=%i pathNo=%i | R=%.15f theta=%.15f phi=%.15f | sun_dist=%i km earth_dist=%i km mars_dist=%i km mars_sun_dist=%i km | min_mars_dist=%i km\n",
                       (t - ts[pathNo]) * UNIT_TIME / (3600.0 * 24.0),
                       i, pathNo, R, theta, phi,
                       (int)sun_distance,
                       (int)earth_distance,
                       (int)mars_distance,
                       (int)mars_orbit,
                       (int)min_mars_distance);
            }
            if (t >= max_duration) {
                printf(".");
                i_final[pathNo] = i;
                break;
            }
            if (i >= max_iter) {
                printf("|");
                i_final[pathNo] = i;
                break;
            }
            /*
            if (earth_distance <= EARTH_RADIUS)
            {
                printf("o");
                i_final[pathNo] = i;
                break;
            }
            */
            if (sun_distance <= SUN_RADIUS)
            {
                printf("*");
                i_final[pathNo] = i;
                break;
            }
            if (mars_distance <= MARS_RADIUS)
            {
                printf("X");
                i_final[pathNo] = i;
                break;
            }
            /*
              if (mars_distance > min_mars_distance)
              {
              printf("-");
              i_final[pathNo] = day;
              score[pathNo] = min_mars_distance;
              break;
              }
            */

            euler_step(h, R, theta, phi, B_R, B_theta, B_phi,
                       R_sun, theta_sun, phi_sun,
                       R_earth, theta_earth, phi_earth,
                       R_mars, theta_mars, phi_mars,
                       &R, &theta, &phi, &B_R, &B_theta, &B_phi);
            t += h;
            i += 1;
        }
        printf("duration=%f i=%i pathNo=%i | R=%.15f theta=%.15f phi=%.15f | sun_dist=%i km earth_dist=%i km mars_dist=%i km mars_sun_dist=%i km | min_mars_dist=%i km\n",
               (t - ts[pathNo]) * UNIT_TIME / (3600.0 * 24.0),
               i, pathNo, R, theta, phi,
               (int)sun_distance,
               (int)earth_distance,
               (int)mars_distance,
               (int)mars_orbit,
               (int)min_mars_distance);
        printf("%6.4f seconds\n", omp_get_wtime() - time);
    }
} // extern "C"
