#include "hip/hip_runtime.h"
#include "constants.h"
#include "coordinate_system.h"
#include "equations_of_motion.h"
#include "equations_of_physics.h"

#include <omp.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

extern "C" {

    void initial_conditions(int nDays,
                            double* days,
                            int nBurndvs,
                            double* burndvs,
                            int nTilts,
                            double* tilts,
                            double altitude,
                            int size_ephemerides,
                            double* earth_R, double* earth_theta, double* earth_phi,
                            double* mars_R, double* mars_theta, double* mars_phi,
                            double* days_out,
                            double* Qs_out,
                            double* Bs_out)
    {
        #pragma omp parallel for
        for (int n = 0; n < nDays * nBurndvs * nTilts; ++n)
        {
            int i = n / (nBurndvs * nTilts);
            int m = n % (nBurndvs * nTilts);
            int j = m / nTilts;
            int k = m % (nTilts);
            int idx = days[i];
            double d = days[i] - idx;
            idx += 1;
            //double R_sun = SUN_R;
            //double theta_sun = SUN_THETA * M_PI / 180.0;
            //double phi_sun = SUN_PHI;
            double R_earth = lerp(earth_R[idx], earth_R[idx + 1], d);
            double theta_earth = lerp(earth_theta[idx], earth_theta[idx + 1], d);
            double phi_earth = lerp(earth_phi[idx], earth_phi[idx + 1], d);

            double x_earth, y_earth, z_earth;
            spherical2cartesian(R_earth, theta_earth, phi_earth, &x_earth, &y_earth, &z_earth);

            double x0_earth, y0_earth, z0_earth, x1_earth, y1_earth, z1_earth;
            spherical2cartesian(earth_R[idx], earth_theta[idx], earth_phi[idx], &x0_earth, &y0_earth, &z0_earth);
            spherical2cartesian(earth_R[idx + 1], earth_theta[idx + 1], earth_phi[idx + 1], &x1_earth, &y1_earth, &z1_earth);

            double vx_earth = (x1_earth - x0_earth) * UNIT_TIME / DAY;
            double vy_earth = (y1_earth - y0_earth) * UNIT_TIME / DAY;
            double vz_earth = (z1_earth - z0_earth) * UNIT_TIME / DAY;
            double norm_v_earth = sqrt(vx_earth * vx_earth +
                                       vy_earth * vy_earth +
                                       vz_earth * vz_earth);
            double vx_earth_unit = vx_earth / norm_v_earth;
            double vy_earth_unit = vy_earth / norm_v_earth;
            double vz_earth_unit = vz_earth / norm_v_earth;

            double x_orbital_plane_unit = y_earth * vz_earth - z_earth * vy_earth;
            double y_orbital_plane_unit = z_earth * vx_earth - x_earth * vz_earth;
            double z_orbital_plane_unit = x_earth * vy_earth - y_earth * vx_earth;
            double norm_orbital_plane_unit = sqrt(x_orbital_plane_unit * x_orbital_plane_unit +
                                             y_orbital_plane_unit * y_orbital_plane_unit +
                                             z_orbital_plane_unit * z_orbital_plane_unit);
            x_orbital_plane_unit /= norm_orbital_plane_unit;
            y_orbital_plane_unit /= norm_orbital_plane_unit;
            z_orbital_plane_unit /= norm_orbital_plane_unit;

            double x_leo_unit = -(y_orbital_plane_unit * vz_earth - z_orbital_plane_unit * vy_earth);
            double y_leo_unit = -(z_orbital_plane_unit * vx_earth - x_orbital_plane_unit * vz_earth);
            double z_leo_unit = -(x_orbital_plane_unit * vy_earth - y_orbital_plane_unit * vx_earth);
            double norm_leo_unit = sqrt(x_leo_unit * x_leo_unit +
                                        y_leo_unit * y_leo_unit +
                                        z_leo_unit * z_leo_unit);
            x_leo_unit /= norm_leo_unit;
            y_leo_unit /= norm_leo_unit;
            z_leo_unit /= norm_leo_unit;

            double x = x_leo_unit * (EARTH_RADIUS + altitude) / UNIT_LENGTH + x_earth;
            double y = y_leo_unit * (EARTH_RADIUS + altitude) / UNIT_LENGTH + y_earth;
            double z = z_leo_unit * (EARTH_RADIUS + altitude) / UNIT_LENGTH + z_earth;

            double e_speed = UNIT_VELOCITY * norm_v_earth;
            double leo_speed = get_circular_orbit_speed(altitude);
            double burn_speed = burndvs[j];

            double vx_leo_unit, vy_leo_unit, vz_leo_unit;
            rotate(vx_earth_unit, vy_earth_unit, vz_earth_unit,
                   x_leo_unit, y_leo_unit, z_leo_unit,
                   &vx_leo_unit, &vy_leo_unit, &vz_leo_unit,
                   tilts[k]);
            double vx_burn_unit, vy_burn_unit, vz_burn_unit;
            rotate(vx_earth_unit, vy_earth_unit, vz_earth_unit,
                   x_orbital_plane_unit, y_orbital_plane_unit, z_orbital_plane_unit,
                   &vx_burn_unit, &vy_burn_unit, &vz_burn_unit,
                   0);
            //double vx_burn = vx_burn_unit * burndvs[j] / UNIT_VELOCITY;
            //double vy_burn = vy_burn_unit * burndvs[j] / UNIT_VELOCITY;
            //double vz_burn = vz_burn_unit * burndvs[j] / UNIT_VELOCITY;
            double vx = (vx_earth_unit * e_speed +
                         vx_leo_unit * leo_speed +
                         vx_leo_unit * burndvs[j]) / UNIT_VELOCITY;
            double vy = (vy_earth_unit * e_speed +
                         vy_leo_unit * leo_speed +
                         vy_leo_unit * burndvs[j]) / UNIT_VELOCITY;

            double vz = (vz_earth_unit * e_speed +
                         vz_leo_unit * leo_speed +
                         vz_leo_unit * burndvs[j]) / UNIT_VELOCITY;
            double c_speed = UNIT_VELOCITY * sqrt(vx * vx + vy * vy + vz * vz);

            double vR_earth, vtheta_earth, vphi_earth;
            velocity_cartesian2spherical(x_earth, y_earth, z_earth, vx_earth, vy_earth, vz_earth, &vR_earth, &vtheta_earth, &vphi_earth);
            double R, theta, phi;
            cartesian2spherical(x, y, z, &R, &theta, &phi);
            double vR, vtheta, vphi;
            velocity_cartesian2spherical(x, y, z, vx, vy, vz, &vR, &vtheta, &vphi);
 
            double earth_distance = UNIT_LENGTH *
                distance(R, theta, phi, R_earth, theta_earth, phi_earth);
            /*
            printf("=========================== %i %i %i %i %.15f===============================\n", n, i, j, k, days[i]);
            printf("e_orbital_plane_cartesian=[%e %e %e]\n", x_orbital_plane_unit, y_orbital_plane_unit, z_orbital_plane_unit);
            printf("e_cartesian=[%f %f %f]\n", x_earth, y_earth, z_earth);
            printf("c_cartesian=[%f %f %f]\n", x, y, z);
            printf("c_e_distance=%f\n", earth_distance);
            printf("----------------------------------------------------\n");
            printf("tilt=%f\n", tilts[k]);
            printf("ev_cartesian=[%f %f %f]\n", vx_earth, vy_earth, vz_earth);
            printf("cv_cartesian=[%f %f %f]\n", vx, vy, vz);
            printf("ev_cartesian_unit=[%f %f %f]\n", vx_earth_unit, vy_earth_unit, vz_earth_unit);
            printf("c_leo_cartesian_unit=[%f %f %f]\n", x_leo_unit, y_leo_unit, z_leo_unit);
            printf("leo_cartesian_unit=[%f %f %f]\n", vx_leo_unit, vy_leo_unit, vz_leo_unit);
            printf("burn_cartesian_unit=[%f %f %f]\n", vx_burn_unit, vy_burn_unit, vz_burn_unit);
            printf("----------------------------------------------------\n");
            printf("e_speed=%f\n", e_speed);
            printf("leo_speed=%f\n", leo_speed);
            printf("burn_speed=%f\n", burn_speed);
            printf("c_speed=%f\n", c_speed);
            printf("----------------------------------------------------\n");
            printf("e_spherical=[%f %f %f]\n", R_earth, theta_earth, phi_earth);
            printf("c_spherical=[%f %f %f]\n", R, theta, phi);
            printf("ev_spherical=[%f %f %f]\n", vR_earth, vtheta_earth, vphi_earth);
            printf("cv_spherical=[%f %f %f]\n", vR, vtheta, vphi);
            printf("====================================================\n");
            */
            days_out[n] = days[i];
            Qs_out[3 * n + 0] = R;
            Qs_out[3 * n + 1] = theta;
            Qs_out[3 * n + 2] = phi;
            Bs_out[3 * n + 0] = get_B_R(vR);
            Bs_out[3 * n + 1] = get_B_theta(R, vtheta);
            Bs_out[3 * n + 2] = get_B_phi(R, theta, vphi);
        }
    }
} // extern "C"
