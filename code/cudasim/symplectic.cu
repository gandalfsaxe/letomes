#include "hip/hip_runtime.h"
#include <omp.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include <math.h>

#include "symplectic.h"
#include "constants.h"

__device__
inline double pdot_denominator1(double x, double y)
{
    return ((x + K) * (x + K) + y * y) * sqrt((x + K) * (x + K) + y * y);
}

__device__
inline double pdot_denominator2(double x, double y)
{
    return ((x - 1 + K) * (x - 1 + K) + y * y) * 
        sqrt((x - 1 + K) * (x - 1 + K) + y * y);
}

// from position and momentum vectors, returns generalized momentum, nondimensionalized
__device__
inline double get_pdot_x(double x, double y, double p_y)
{
    return p_y - ((1 - K) * (x + K)) / pdot_denominator1(x, y) + 
        K * (x - 1 + K) / pdot_denominator2(x, y);
}

__device__
inline double get_pdot_y(double x, double y, double p_x)
{
    return -p_x - (1 - K) * y / pdot_denominator1(x, y) - 
        K * y / pdot_denominator2(x, y);
}

__device__
inline double get_v_x(double y, double p_x)
{
    return p_x + y;
}

__device__
inline double get_v_y(double x, double p_y)
{
    return p_y - x;
}

// Runs symplectic adaptive euler-verlet algorithm
// All values are with nondimensionalized units
__device__
void symplectic(double x0,
                double y0,
                double p0_x,
                double p0_y,
                double maxDuration,
                double maxIter,
                double* score,
                bool* success
                )
{
    //success[0] = 0
    double h = h_DEFAULT;
    double h_min = h_MIN;
    double t = 0;  // total elapsed time
    double x = x0;
    double y = y0;
    double p_x = p0_x;
    double p_y = p0_y;
    //path_storage = []
    //path_storage.append([x, y, p_x, p_y, h])
    double smallest_distance = 1e6;
    double Dv = 0;
    int iteration_count = 0;

    double target_orbital_radius = LLO_RADIUS;
    double target_orbital_velocity = LLO_VELOCITY;
    double target_position_x = LUNAR_POSITION_X;
    double target_position_y = 0;
    double target_orbital_velocity_nondim = target_orbital_velocity / UNIT_VELOCITY;

    double earth_orbital_radius = LEO_RADIUS;
    double earth_orbital_velocity = LEO_VELOCITY;
    double earth_position_x = EARTH_POSITION_X;
    double earth_position_y = 0;
    double earth_celestial_radius = EARTH_RADIUS;
    double orbital_radius_lower_bound =
        (target_orbital_radius - ORBITAL_TOLERANCE) / UNIT_LENGTH;
    double orbital_radius_upper_bound = 
        (target_orbital_radius + ORBITAL_TOLERANCE) / UNIT_LENGTH;
    float too_far_away = 4 / UNIT_LENGTH;

    while (t < maxDuration)
    {
        if (iteration_count > maxIter)
        {
            // printf("exceeded max iterations, stranded in space!\n");
            printf(".");
            success[0] = false;
            score[0] = smallest_distance;
            return; //return path_storage
        }

        //========= EULER STEP ==========
        // Take a single time step of the symplectic Euler algorithm
        double x_euler, y_euler; //, p_x_euler, p_y_euler;
        {
            //printf("input = %f %f %f %f %f\n", h, x, y, p_x, p_y);
            // Step 1
            double v_x = get_v_x(y, p_x);
            double x_ = (x + (v_x + p_y * h) * h) / (1.0 + h * h);
            // Step 2
            double v_y = get_v_y(x_, p_y);
            double y_ = y + v_y * h;
            // Step 3
            //double pdot_x = get_pdot_x(x_, y_, p_y);
            //double pdot_y = get_pdot_y(x_, y_, p_x);
            //double p_x_ = p_x + pdot_x * h;
            //double p_y_ = p_y + pdot_y * h;
            x_euler = x_;
            y_euler = y_;
            //p_x_euler = p_x_;
            //p_y_euler = p_y_;
            //printf("%.15f %.15f %.15f %.15f\n", x_, v_x, pdot_x, p_x_);
            //printf("%.15f %.15f %.15f %.15f\n", y_, v_y, pdot_y, p_y_);
        }
        //printf("euler: %.15f %.15f %.15f %.15f\n", x_euler, y_euler, p_x_euler, p_y_euler);

        //========= VERLET STEP ==========
        // Takes a half step, then another half step in the symplectic Verlet algorithm"""
        double x_verlet, y_verlet, p_x_verlet, p_y_verlet;
        {
            double hh = 0.5 * h;
            double denominator = 1.0 / (1.0 + hh * hh);
            // Step 1
            double v_x = get_v_x(y, p_x);
            double x_ = (x + (v_x + p_y * hh) * hh) * denominator;
            // Step 2
            double v_y = get_v_y(x_, p_y);
            double y_ = y + v_y * hh;
            // Step 2
            double pdot_x = get_pdot_x(x_, y_, p_y);
            double pdot_y = get_pdot_y(x_, y_, p_x);
            double p_x_ = (p_x + (2.0 * pdot_x + (2 * pdot_y + p_x) * hh) * hh) * denominator;
            double p_y_ = p_y + (pdot_y + get_pdot_y(x_, y_, p_x_)) * hh;
            // TODO: mixed, what's correct? Derive theory
            // Step 3
            v_x = get_v_x(y_, p_x_);
            v_y = get_v_y(x_, p_y_);
            x_ += v_x * hh;
            y_ += v_y * hh;
            x_verlet = x_;
            y_verlet = y_;
            p_x_verlet = p_x_;
            p_y_verlet = p_y_;
        }
        //printf("verlet: %.15f %.15f %.15f %.15f\n", x_verlet, y_verlet, p_x_verlet, p_y_verlet);

        double err = sqrt((
            (x_verlet - x_euler) * (x_verlet - x_euler) + 
            (y_verlet - y_euler) * (y_verlet - y_euler)) / 
            (x_verlet * x_verlet + y_verlet * y_verlet));
        //printf("err = %.15g\n", err);

        if (err < STEP_ERROR_TOLERANCE || h <= h_min)
        {
            iteration_count += 1;
            x = x_verlet;
            y = y_verlet;
            p_x = p_x_verlet;
            p_y = p_y_verlet;
            t += h;
            h = max(h_min, h * max(0.1, 0.8 * sqrt(STEP_ERROR_TOLERANCE / err)));
            //printf("accept step h=%.15lf, err=%.15lf, x = %.15lf, t = %.15lf\n", h, err, x, t);
            // Accept the step only if the weighted error is no more than the
            // tolerance tol.  Estimate an h that will yield an error of tol on
            // the next step and use 0.8 of this value to avoid failures.
        }
        else
        {
            h = max(h_min, h / 2);
            //printf("deny step h=%.15lf, err=%.15lf, x = %.15lf, t = %.15lf\n", h, err, x, t);
        }

        // Are we nearly there yet? (calculate distance)
        double target_distance_x = x - target_position_x;
        double target_distance_y = y - target_position_y;
        double target_distance = sqrt(target_distance_x * target_distance_x + target_distance_y * target_distance_y);
        if (target_distance > too_far_away)
        {
            // printf("we are way too far away, stranded in space!\n");
            printf("|");
            success[0] = false;
            score[0] = smallest_distance;
            return; //path_storage
        }
        smallest_distance = min(smallest_distance, target_distance);

        // For real though, are we there yet? (did we actually hit?)
        if (smallest_distance >= orbital_radius_lower_bound &&
            smallest_distance <= orbital_radius_upper_bound)
        {
            // SUCCESS! We are in orbit range
            // current velocity vector
            double v_x = p_x + y;
            double v_y = p_y - x;

            // We adjust our velocity so the spacecraft enters a closed circular orbit.
            // We treat target_distance as a vector from spacecraft to target

            // project velocity vector onto radial direction unit-vector. This is what we
            // want to subtract from the velocity vector to obtain the tangental component (closed circular orbit)
            double v_radial = (v_x * target_distance_x + v_y * target_distance_y) 
                / target_distance;

            // phi is the angle of the radial vector
            double cos_phi = target_distance_x / target_distance;
            double sin_phi = target_distance_y / target_distance;
            // project radial velocity component to x and y axes.
            v_x = v_x - v_radial * cos_phi;
            v_y = v_y - v_radial * sin_phi;
            double v_magnitude = sqrt(v_x * v_x + v_y * v_y);

            // Delta-V for the maneuver
            Dv = sqrt(v_radial * v_radial + 
                      (v_magnitude - target_orbital_velocity_nondim) * (v_magnitude - target_orbital_velocity_nondim));
            //printf("SUCCESS! duration=%8.6g, Dv=%17.15g, iteration_count=%-10i\n", t * UNIT_TIME, Dv, iteration_count);
            printf("O");
            success[0] = true;
            score[0] = Dv;
            return; // path_storage
        }

        //path_storage.append([x, y, p_x, p_y, h])
        //printf("[x, y, p_x, p_y, h] = [%.15g, %.15g, %.15g, %.15g, %.15g]\n", x, y, p_x, p_y, h);

        // check if we somehow accidentally struck the earth (whoops)
        double earth_distance_sqr =
            (x - earth_position_x) * (x - earth_position_x) + 
            (y - earth_position_y) * (y - earth_position_y);
        // not necessarily a crash, but we don't want paths that take us to such risky territories
        double critical_distance = (earth_celestial_radius / UNIT_LENGTH) * (earth_celestial_radius / UNIT_LENGTH);
        if (earth_distance_sqr < critical_distance)
        {
            // printf("Anga crashed into the earth!\n");
            printf("X");
            success[0] = false;
            score[0] = smallest_distance;
            return; // path_storage
        }

    }
    //printf("exceeded max duration!\n");
    //printf("smallest distance=%g, orbital_radius_lower_bound=%g, orbital_radius_upper_bound=%g\n", smallest_distance, orbital_radius_lower_bound, orbital_radius_upper_bound);
    success[0] = false;
    score[0] = smallest_distance;
    return; // path_storage
}
